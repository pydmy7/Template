#include <cstdio>

#include <hip/hip_runtime.h>

// CUDA kernel function
__global__ void helloFromGPU() {
    [[maybe_unused]] int xxx = threadIdx.x;
}

int main() {
    printf("Hello from CPU!\n");

    // Launch kernel with 4 threads
    helloFromGPU<<<1, 4>>>();

    // Wait for GPU to finish
    hipDeviceSynchronize();

    return 0;
}
