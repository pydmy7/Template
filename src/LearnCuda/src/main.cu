#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>

#include "cudapp.cuh"

__global__ void kernel() {
    printf("Hello, world!\n");
}

int main() {
    kernel<<<1, 3>>>();
    hipDeviceSynchronize();
    return 0;
}
